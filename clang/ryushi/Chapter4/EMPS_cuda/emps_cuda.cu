#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __host__
#define __shared__
#endif

#define IN_FILE "../mk_particle/dambreak.prof"
#define PCL_DST 0.02
#define MIN_X  (0.0 - PCL_DST*3)
#define MIN_Y  (0.0 - PCL_DST*3)
#define MIN_Z  (0.0 - PCL_DST*3)
#define MAX_X  (1.0 + PCL_DST*3)
#define MAX_Y  (0.2 + PCL_DST*3)
#define MAX_Z  (0.6 + PCL_DST*30)

#define GST -1
#define FLD 0
#define WLL  1
#define NUM_TYP  2
#define DNS_FLD 1000
#define DNS_WLL 1000
#define DT 0.0005
#define FIN_TIM 1.0
#define SND 22.0
#define OPT_FQC 100
#define KNM_VSC 0.000001
#define DIM 3
#define CRT_NUM 0.1
#define COL_RAT 0.2
#define DST_LMT_RAT 0.9
#define G_X 0.0
#define G_Y 0.0
#define G_Z -9.8
#define WEI(dst, re) 		((re/dst) - 1.0)

FILE* fp;
char filename[256];
int iLP,iF;
double TIM;
int nP;
double *Acc,*Pos,*Vel,*Prs,*pav;
int *Typ;
double r,r2;
double DB,DB2,DBinv;
int nBx,nBy,nBz,nBxy,nBxyz;
double n0,lmd,A1,A2,A3,rlim,rlim2,COL;
double Dns[NUM_TYP],invDns[NUM_TYP];

void ChkPcl(int i){
	if(Typ[i] != GST){
	if(	Pos[i*3  ]>MAX_X || Pos[i*3  ]<MIN_X ||
		Pos[i*3+1]>MAX_Y || Pos[i*3+1]<MIN_Y ||
		Pos[i*3+2]>MAX_Z || Pos[i*3+2]<MIN_Z)
	{
		Typ[i] = GST;
		Prs[i]=Vel[i*3]=Vel[i*3+1]=Vel[i*3+2]=0.0;
	}}
}

void RdDat(void) {
	fp = fopen(IN_FILE, "r");
	fscanf(fp,"%d",&nP);
	printf("nP: %d\n",nP);
	Acc = (double*)malloc(sizeof(double)*nP*3);
	Pos = (double*)malloc(sizeof(double)*nP*3);
	Vel = (double*)malloc(sizeof(double)*nP*3);
	Prs = (double*)malloc(sizeof(double)*nP);
	pav = (double*)malloc(sizeof(double)*nP);
	Typ = (int*)malloc(sizeof(int)*nP);
	for(int i=0;i<nP;i++) {
		int a[2];
		double b[8];
		fscanf(fp," %d %d %lf %lf %lf %lf %lf %lf %lf %lf",&a[0],&a[1],&b[0],&b[1],&b[2],&b[3],&b[4],&b[5],&b[6],&b[7]);
		Typ[i]=a[1];
		Pos[i*3]=b[0];	Pos[i*3+1]=b[1];	Pos[i*3+2]=b[2];
		Vel[i*3]=b[3];	Vel[i*3+1]=b[4];	Vel[i*3+2]=b[5];
		Prs[i]=b[6];		pav[i]=b[7];
	}
	fclose(fp);
	for(int i=0;i<nP;i++) {ChkPcl(i);}
	for(int i=0;i<nP*3;i++) {Acc[i]=0.0;}
}

void WrtDat(void) {
	char outout_filename[256];
	sprintf(outout_filename, "output%05d.prof",iF);
	fp = fopen(outout_filename, "w");
	fprintf(fp,"%d\n",nP);
	for(int i=0;i<nP;i++) {
		int a[2];
		double b[8];
		a[0]=i;	a[1]=Typ[i];
		b[0]=Pos[i*3];	b[1]=Pos[i*3+1];	b[2]=Pos[i*3+2];
		b[3]=Vel[i*3];	b[4]=Vel[i*3+1];	b[5]=Vel[i*3+2];
		b[6]=Prs[i];		b[7]=pav[i]/OPT_FQC;
		fprintf(fp," %d %d %lf %lf %lf %lf %lf %lf %lf %lf\n",a[0],a[1],b[0],b[1],b[2],b[3],b[4],b[5],b[6],b[7]);
		pav[i]=0.0;
	}
	fclose(fp);
	iF++;
}

void AlcBkt(void) {
	r = PCL_DST*2.1;		r2 = r*r;
	DB = r*(1.0+CRT_NUM);	DB2 = DB*DB;		DBinv = 1.0/DB;
	nBx = (int)((MAX_X - MIN_X)*DBinv) + 3;
	nBy = (int)((MAX_Y - MIN_Y)*DBinv) + 3;
	nBz = (int)((MAX_Z - MIN_Z)*DBinv) + 3;
	nBxy = nBx*nBy;
	nBxyz = nBx*nBy*nBz;
	printf("nBx:%d  nBy:%d  nBz:%d  nBxy:%d  nBxyz:%d\n",nBx,nBy,nBz,nBxy,nBxyz);
}

void SetPara(void){
	n0 = lmd =0.0;
	for(int ix= -4;ix<5;ix++){
	for(int iy= -4;iy<5;iy++){
	for(int iz= -4;iz<5;iz++){
		double x = PCL_DST* (double)ix;
		double y = PCL_DST* (double)iy;
		double z = PCL_DST* (double)iz;
		double dst2 = x*x+y*y+z*z;
		if(dst2 <= r2){
			if(dst2==0.0)continue;
			double dst = sqrt(dst2);
			n0 += WEI(dst, r);
			lmd += dst2 * WEI(dst, r);
		}
	}}}
	lmd = lmd/n0;
	A1 = 2.0*KNM_VSC*DIM/n0/lmd;
	A2 = SND*SND/n0;
	A3 = -DIM/n0;
	Dns[FLD]=DNS_FLD;			Dns[WLL]=DNS_WLL;
	invDns[FLD]=1.0/DNS_FLD;	invDns[WLL]=1.0/DNS_WLL;
	rlim = PCL_DST * DST_LMT_RAT;	rlim2 = rlim*rlim;
	COL = 1.0 + COL_RAT;
	iLP=iF=0;
	TIM=0.0;
}


int BLOCKS,TOTAL_THREADS;
#define THREADS 128

#define ERR_CHK(func){ checkCudaErrors(func); }
#define ERR_KNL(func){ func; getLastCudaError(#func); }

__global__ void d_initialize_double_array(int n, double *d_array, double a){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){d_array[i] = a;}
}

__global__ void d_initialize_int_array(int n, int *i_array, int a){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){i_array[i] = a;}
}

__global__ void d_add_double_array(int n, double *d_array1, double *d_array0){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){d_array1[i] += d_array0[i];}
}

double *d_Acc,*d_Pos,*d_Vel,*d_Prs,*d_pav;
int *d_Typ;
int *d_bfst,*d_blst,*d_nxt;
double *d_Dns,*d_invDns;

__device__ void d_ChkPcl(int i, int d_nP,	int *d_Typ, double* d_Pos, double* d_Vel, double* d_Acc, double* d_Prs)
{
	if(d_Typ[i] != GST){
	if(	d_Pos[i*3  ]>MAX_X || d_Pos[i*3  ]<MIN_X ||
		d_Pos[i*3+1]>MAX_Y || d_Pos[i*3+1]<MIN_Y ||
		d_Pos[i*3+2]>MAX_Z || d_Pos[i*3+2]<MIN_Z)
	{
		d_Typ[i] = GST;
		d_Prs[i]=d_Vel[i*3]=d_Vel[i*3+1]=d_Vel[i*3+2]=0.0;
	}}
}

__global__ void d_MkBkt(int d_nP, int d_nBx, int d_nBxy, int d_nBxyz, double d_DBinv,
		int* d_bfst, int* d_blst, int* d_nxt,
		int *d_Typ, double* d_Pos)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
//	int i = blockIdx.x + threadIdx.x*gridDim.x;
	if(i<d_nP){
		if(d_Typ[i] != GST){
			int ix = (int)((d_Pos[i*3  ] - MIN_X)*d_DBinv) +1;
			int iy = (int)((d_Pos[i*3+1] - MIN_Y)*d_DBinv) +1;
			int iz = (int)((d_Pos[i*3+2] - MIN_Z)*d_DBinv) +1;

			int ib = iz*d_nBxy + iy*d_nBx + ix;
			int j = atomicExch(&d_blst[ib],i);
			if(j == -1){	d_bfst[ib] = i;	}
			else{				d_nxt[j] = i;}
		}
	}
}

__global__ void d_VscTrm(int d_nP, int d_nBx, int d_nBxy, int d_nBxyz, double d_DBinv,
		int* d_bfst, int* d_blst, int* d_nxt,
		int *d_Typ, double* d_Pos, double* d_Vel, double* d_Acc, double d_r, double d_A1)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<d_nP){
	if(d_Typ[i] == FLD){
		double Acc_x = 0.0;			double Acc_y = 0.0;			double Acc_z = 0.0;
		double pos_ix = d_Pos[i*3  ];	double pos_iy = d_Pos[i*3+1];	double pos_iz = d_Pos[i*3+2];
		double vec_ix = d_Vel[i*3  ];	double vec_iy = d_Vel[i*3+1];	double vec_iz = d_Vel[i*3+2];
		int ix = (int)((pos_ix - MIN_X)*d_DBinv) +1;
		int iy = (int)((pos_iy - MIN_Y)*d_DBinv) +1;
		int iz = (int)((pos_iz - MIN_Z)*d_DBinv) +1;
		for(int jz=iz-1;jz<=iz+1;jz++){
		for(int jy=iy-1;jy<=iy+1;jy++){
		for(int jx=ix-1;jx<=ix+1;jx++){
			int jb = jz*d_nBxy + jy*d_nBx + jx;
			int j = d_bfst[jb];
			if(j == -1) continue;
			for(;;){
				double v0 = d_Pos[j*3  ] - pos_ix;
				double v1 = d_Pos[j*3+1] - pos_iy;
				double v2 = d_Pos[j*3+2] - pos_iz;
				double dst2 = v0*v0+v1*v1+v2*v2;
				if(dst2<d_r*d_r){
				if(j!=i && d_Typ[j]!=GST){
					double dst = sqrt(dst2);
					double w =  WEI(dst, d_r);
					Acc_x +=(d_Vel[j*3  ]-vec_ix)*w;
					Acc_y +=(d_Vel[j*3+1]-vec_iy)*w;
					Acc_z +=(d_Vel[j*3+2]-vec_iz)*w;
				}}
				j = d_nxt[j];
				if(j==-1) break;
			}
		}}}
		d_Acc[i*3  ]=Acc_x*d_A1 + G_X;
		d_Acc[i*3+1]=Acc_y*d_A1 + G_Y;
		d_Acc[i*3+2]=Acc_z*d_A1 + G_Z;
	}}
}

__global__ void d_UpPcl1(	int d_nP, int *d_Typ, double* d_Pos, double* d_Vel, double* d_Acc, double* d_Prs)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<d_nP){
		if(d_Typ[i] == FLD){
			d_Vel[i*3  ] +=d_Acc[i*3  ]*DT;	d_Vel[i*3+1] +=d_Acc[i*3+1]*DT;	d_Vel[i*3+2] +=d_Acc[i*3+2]*DT;
			d_Pos[i*3  ] +=d_Vel[i*3  ]*DT;		d_Pos[i*3+1] +=d_Vel[i*3+1]*DT;		d_Pos[i*3+2] +=d_Vel[i*3+2]*DT;
			d_Acc[i*3]=d_Acc[i*3+1]=d_Acc[i*3+2]=0.0;
			d_ChkPcl(i, d_nP, d_Typ, d_Pos, d_Vel, d_Acc, d_Prs);
		}
	}
}

__global__ void d_ChkCol(int d_nP, int d_nBx, int d_nBxy, int d_nBxyz, double d_DBinv,
		int* d_bfst, int* d_blst, int* d_nxt,
		int *d_Typ, double* d_Pos, double* d_Vel, double* d_Acc, double *d_Dns , double d_rlim2, double d_COL)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<d_nP){
	if(d_Typ[i] == FLD){
		double mi = d_Dns[d_Typ[i]];
		double pos_ix = d_Pos[i*3  ];	double pos_iy = d_Pos[i*3+1];	double pos_iz = d_Pos[i*3+2];
		double vec_ix = d_Vel[i*3  ];	double vec_iy = d_Vel[i*3+1];	double vec_iz = d_Vel[i*3+2];
		double vec_ix2 = d_Vel[i*3  ];	double vec_iy2 = d_Vel[i*3+1];	double vec_iz2 = d_Vel[i*3+2];
		int ix = (int)((pos_ix - MIN_X)*d_DBinv) +1;
		int iy = (int)((pos_iy - MIN_Y)*d_DBinv) +1;
		int iz = (int)((pos_iz - MIN_Z)*d_DBinv) +1;
		for(int jz=iz-1;jz<=iz+1;jz++){
		for(int jy=iy-1;jy<=iy+1;jy++){
		for(int jx=ix-1;jx<=ix+1;jx++){
			int jb = jz*d_nBxy + jy*d_nBx + jx;
			int j = d_bfst[jb];
			if(j == -1) continue;
			for(;;){
				double v0 = d_Pos[j*3  ] - pos_ix;
				double v1 = d_Pos[j*3+1] - pos_iy;
				double v2 = d_Pos[j*3+2] - pos_iz;
				double dst2 = v0*v0+v1*v1+v2*v2;
				if(dst2<d_rlim2){
				if(j!=i && d_Typ[j]!=GST){
					double fDT = (vec_ix-d_Vel[j*3  ])*v0+(vec_iy-d_Vel[j*3+1])*v1+(vec_iz-d_Vel[j*3+2])*v2;
					if(fDT > 0.0){
						double mj = d_Dns[d_Typ[j]];
						fDT *= d_COL*mj/(mi+mj)/dst2;
						vec_ix2 -= v0*fDT;		vec_iy2 -= v1*fDT;		vec_iz2 -= v2*fDT;
					}
				}}
				j = d_nxt[j];
				if(j==-1) break;
			}
		}}}
		d_Acc[i*3  ]=vec_ix2;	d_Acc[i*3+1]=vec_iy2;	d_Acc[i*3+2]=vec_iz2;
	}}
}
__global__ void d_MkPrs(int d_nP, int d_nBx, int d_nBxy, int d_nBxyz, double d_DBinv,
		int* d_bfst, int* d_blst, int* d_nxt,
		int *d_Typ, double* d_Pos, double* d_Prs, double *d_Dns ,double d_r, double d_n0, double d_A2)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<d_nP){
	if(d_Typ[i] != GST){
		double pos_ix = d_Pos[i*3  ];	double pos_iy = d_Pos[i*3+1];	double pos_iz = d_Pos[i*3+2];
		double ni = 0.0;
		int ix = (int)((pos_ix - MIN_X)*d_DBinv) +1;
		int iy = (int)((pos_iy - MIN_Y)*d_DBinv) +1;
		int iz = (int)((pos_iz - MIN_Z)*d_DBinv) +1;
		for(int jz=iz-1;jz<=iz+1;jz++){
		for(int jy=iy-1;jy<=iy+1;jy++){
		for(int jx=ix-1;jx<=ix+1;jx++){
			int jb = jz*d_nBxy + jy*d_nBx + jx;
			int j = d_bfst[jb];
			if(j == -1) continue;
			for(;;){
				double v0 = d_Pos[j*3  ] - pos_ix;
				double v1 = d_Pos[j*3+1] - pos_iy;
				double v2 = d_Pos[j*3+2] - pos_iz;
				double dst2 = v0*v0+v1*v1+v2*v2;
				if(dst2<d_r*d_r){
				if(j!=i && d_Typ[j]!=GST){
					double dst = sqrt(dst2);
					double w =  WEI(dst, d_r);
					ni += w;
				}}
				j = d_nxt[j];
				if(j==-1) break;
			}
		}}}
		double mi = d_Dns[d_Typ[i]];
		double pressure = (ni > d_n0)*(ni - d_n0) * d_A2 * mi;
		d_Prs[i] = pressure;
	}}
}

__global__ void d_PrsGrdTrm(int d_nP, int d_nBx, int d_nBxy, int d_nBxyz, double d_DBinv,
		int* d_bfst, int* d_blst, int* d_nxt,
		int *d_Typ, double* d_Pos, double* d_Acc, double* d_Prs, double *d_invDns, double d_r, double d_A3)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<d_nP){
	if(d_Typ[i] == FLD){
		double Acc_x = 0.0;			double Acc_y = 0.0;			double Acc_z = 0.0;
		double pos_ix = d_Pos[i*3  ];	double pos_iy = d_Pos[i*3+1];	double pos_iz = d_Pos[i*3+2];
		double pre_min = d_Prs[i];
		int ix = (int)((pos_ix - MIN_X)*d_DBinv) +1;
		int iy = (int)((pos_iy - MIN_Y)*d_DBinv) +1;
		int iz = (int)((pos_iz - MIN_Z)*d_DBinv) +1;
		for(int jz=iz-1;jz<=iz+1;jz++){
		for(int jy=iy-1;jy<=iy+1;jy++){
		for(int jx=ix-1;jx<=ix+1;jx++){
			int jb = jz*d_nBxy + jy*d_nBx + jx;
			int j = d_bfst[jb];
			if(j == -1) continue;
			for(;;){
				double v0 = d_Pos[j*3  ] - pos_ix;
				double v1 = d_Pos[j*3+1] - pos_iy;
				double v2 = d_Pos[j*3+2] - pos_iz;
				double dst2 = v0*v0+v1*v1+v2*v2;
				if(dst2<d_r*d_r){
				if(j!=i && d_Typ[j]!=GST){
					if(pre_min > d_Prs[j])pre_min = d_Prs[j];
				}}
				j = d_nxt[j];
				if(j==-1) break;
			}
		}}}
		for(int jz=iz-1;jz<=iz+1;jz++){
		for(int jy=iy-1;jy<=iy+1;jy++){
		for(int jx=ix-1;jx<=ix+1;jx++){
			int jb = jz*d_nBxy + jy*d_nBx + jx;
			int j = d_bfst[jb];
			if(j == -1) continue;
			for(;;){
				double v0 = d_Pos[j*3  ] - pos_ix;
				double v1 = d_Pos[j*3+1] - pos_iy;
				double v2 = d_Pos[j*3+2] - pos_iz;
				double dst2 = v0*v0+v1*v1+v2*v2;
				if(dst2<d_r*d_r){
				if(j!=i && d_Typ[j]!=GST){
					double dst = sqrt(dst2);
					double w =  WEI(dst, d_r);
					w *= (d_Prs[j] - pre_min)/dst2;
					Acc_x += v0*w;	Acc_y += v1*w;	Acc_z += v2*w;
				}}
				j = d_nxt[j];
				if(j==-1) break;
			}
		}}}
		d_Acc[i*3  ]=Acc_x*d_invDns[FLD]*d_A3;
		d_Acc[i*3+1]=Acc_y*d_invDns[FLD]*d_A3;
		d_Acc[i*3+2]=Acc_z*d_invDns[FLD]*d_A3;
	}}
}

__global__ void d_UpPcl2(int d_nP,	int *d_Typ, double* d_Pos, double* d_Vel, double* d_Acc, double* d_Prs)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<d_nP){
		if(d_Typ[i] == FLD){
			d_Vel[i*3  ] +=d_Acc[i*3  ]*DT;		d_Vel[i*3+1] +=d_Acc[i*3+1]*DT;		d_Vel[i*3+2] +=d_Acc[i*3+2]*DT;
			d_Pos[i*3  ] +=d_Acc[i*3  ]*DT*DT;	d_Pos[i*3+1] +=d_Acc[i*3+1]*DT*DT;	d_Pos[i*3+2] +=d_Acc[i*3+2]*DT*DT;
			d_Acc[i*3]=d_Acc[i*3+1]=d_Acc[i*3+2]=0.0;
			d_ChkPcl(i, d_nP, d_Typ, d_Pos, d_Vel, d_Acc, d_Prs);
		}
	}
}

void ClcEMPS_cuda(void){

	dim3 threads(THREADS, 1, 1);
	TOTAL_THREADS = nBxyz;	BLOCKS = TOTAL_THREADS/THREADS+1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = nP;	BLOCKS = TOTAL_THREADS/THREADS+1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	TOTAL_THREADS = nP*3;	BLOCKS = TOTAL_THREADS/THREADS+1;
	dim3 blocks_nP3(BLOCKS, 1, 1);

	while(1){
		if(iLP%OPT_FQC == 0 ){
			ERR_CHK(hipMemcpy(Typ, d_Typ, sizeof(int)*nP, hipMemcpyDeviceToHost));
			ERR_CHK(hipMemcpy(Pos, d_Pos, sizeof(double)*nP*3, hipMemcpyDeviceToHost));
			ERR_CHK(hipMemcpy(Vel, d_Vel, sizeof(double)*nP*3, hipMemcpyDeviceToHost));
			ERR_CHK(hipMemcpy(Prs, d_Prs, sizeof(double)*nP, hipMemcpyDeviceToHost));
			ERR_CHK(hipMemcpy(pav, d_pav, sizeof(double)*nP, hipMemcpyDeviceToHost));
			ERR_KNL((d_initialize_double_array<<<blocks_nP, threads>>>(nP, d_pav, 0.0)));
			WrtDat();

			int p_num=0;
			for(int i=0;i<nP;i++){if(Typ[i] != GST)p_num++;}
			printf("%5d th TIM: %lf / p_num: %d\n", iLP,TIM,p_num);

			if(TIM >= FIN_TIM ){
				break;
			}
		}

		ERR_KNL((d_initialize_int_array<<<blocks_nBxyz, threads>>>(nBxyz, d_bfst, -1)));
		ERR_KNL((d_initialize_int_array<<<blocks_nBxyz, threads>>>(nBxyz, d_blst, -1)));
		ERR_KNL((d_initialize_int_array<<<blocks_nP, threads>>>(nP, d_nxt, -1)));
		ERR_KNL((d_MkBkt<<<blocks_nP, threads>>>(nP, nBx, nBxy, nBxyz, DBinv,
				d_bfst, d_blst, d_nxt, d_Typ, d_Pos)));

		ERR_KNL((d_VscTrm<<<blocks_nP, threads>>>(nP, nBx, nBxy, nBxyz, DBinv,
				d_bfst, d_blst, d_nxt, d_Typ, d_Pos, d_Vel, d_Acc, r, A1)));
		ERR_KNL((d_UpPcl1<<<blocks_nP, threads>>>(nP, d_Typ, d_Pos, d_Vel, d_Acc, d_Prs)));

		ERR_KNL((d_ChkCol<<<blocks_nP, threads>>>(nP, nBx, nBxy, nBxyz, DBinv,
				d_bfst, d_blst, d_nxt, d_Typ, d_Pos, d_Vel, d_Acc, d_Dns, rlim2, COL)));
		ERR_CHK(hipMemcpy(d_Vel, d_Acc, sizeof(double)*nP*3, hipMemcpyDeviceToDevice));


		ERR_KNL((d_MkPrs<<<blocks_nP, threads>>>(nP, nBx, nBxy, nBxyz, DBinv,
				d_bfst, d_blst, d_nxt, d_Typ, d_Pos, d_Prs, d_Dns, r, n0, A2)));

		ERR_KNL((d_PrsGrdTrm<<<blocks_nP, threads>>>(nP, nBx, nBxy, nBxyz, DBinv,
				d_bfst, d_blst, d_nxt, d_Typ, d_Pos, d_Acc, d_Prs, d_invDns, r, A3)));
		ERR_KNL((d_UpPcl2<<<blocks_nP, threads>>>(nP, d_Typ, d_Pos, d_Vel, d_Acc, d_Prs)));

		ERR_KNL((d_MkPrs<<<blocks_nP, threads>>>(nP, nBx, nBxy, nBxyz, DBinv,
				d_bfst, d_blst, d_nxt, d_Typ, d_Pos, d_Prs, d_Dns, r, n0, A2)));
		ERR_KNL((d_add_double_array<<<blocks_nP, threads>>>(nP,d_pav,d_Prs)));

		iLP++;
		TIM += DT;
	}
}
#include <sys/time.h>
double get_dtime(void){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return ((double)(tv.tv_sec) + (double)(tv.tv_usec) * 0.000001);
}
int main( int argc, char** argv)
{
	ERR_CHK(hipSetDevice(0));

	printf("start emps_cuda.\n");
	RdDat();

	ERR_CHK(hipMalloc( (void**) &d_Typ, sizeof(int)*nP ));
	ERR_CHK(hipMalloc( (void**) &d_Acc, sizeof(double)*nP*3 ));
	ERR_CHK(hipMalloc( (void**) &d_Pos, sizeof(double)*nP*3 ));
	ERR_CHK(hipMalloc( (void**) &d_Vel, sizeof(double)*nP*3 ));
	ERR_CHK(hipMalloc( (void**) &d_Prs, sizeof(double)*nP ));
	ERR_CHK(hipMalloc( (void**) &d_pav, sizeof(double)*nP ));

	ERR_CHK(hipMemcpy(d_Typ, 	Typ,	sizeof(int)*nP, 		hipMemcpyHostToDevice));
	ERR_CHK(hipMemcpy(d_Acc, 	Acc, 	sizeof(double)*nP*3, hipMemcpyHostToDevice));
	ERR_CHK(hipMemcpy(d_Pos, 	Pos,	sizeof(double)*nP*3, hipMemcpyHostToDevice));
	ERR_CHK(hipMemcpy(d_Vel, 	Vel,	sizeof(double)*nP*3,	hipMemcpyHostToDevice));
	ERR_CHK(hipMemcpy(d_Prs, 	Prs,	sizeof(double)*nP, 	hipMemcpyHostToDevice));
	ERR_CHK(hipMemcpy(d_pav, 	pav,	sizeof(double)*nP, 	hipMemcpyHostToDevice));

	AlcBkt();

	ERR_CHK(hipMalloc( (void**) &d_bfst, sizeof(int)*nBxyz ));
	ERR_CHK(hipMalloc( (void**) &d_blst, sizeof(int)*nBxyz ));
	ERR_CHK(hipMalloc( (void**) &d_nxt, sizeof(int)*nP ));

	SetPara();

	ERR_CHK(hipMalloc( (void**) &d_Dns,		sizeof(double)*2 ));
	ERR_CHK(hipMalloc( (void**) &d_invDns, 	sizeof(double)*2 ));
	ERR_CHK(hipMemcpy(d_Dns, 	Dns,		sizeof(double)*2, hipMemcpyHostToDevice));
	ERR_CHK(hipMemcpy(d_invDns,invDns,	sizeof(double)*2, hipMemcpyHostToDevice));

	double timer_sta = get_dtime();

	ClcEMPS_cuda();

	double timer_end = get_dtime();
	printf("Total        : %13.6lf sec\n",timer_end -timer_sta);

	ERR_CHK(hipFree(d_Typ));	ERR_CHK(hipFree(d_Acc));	ERR_CHK(hipFree(d_Pos));
	ERR_CHK(hipFree(d_Vel));	ERR_CHK(hipFree(d_Prs));	ERR_CHK(hipFree(d_pav));
	ERR_CHK(hipFree(d_bfst));	ERR_CHK(hipFree(d_blst));	ERR_CHK(hipFree(d_nxt));
	free(Acc);	free(Pos);	free(Vel);	free(Prs);	free(pav);	free(Typ);
	printf("end emps_cuda.\n");

	ERR_CHK(hipDeviceReset());
	return 0;
}
